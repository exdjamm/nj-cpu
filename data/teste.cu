#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

typedef struct{
    int size;
    float *elements;
} Vector;

typedef struct{
    int n;
    float *elements;
} Matrix;


// Matrix
__device__ float d_get_matrix_position(Matrix D, int i, int j);
// float h_get_position(Matrix D, int i, int j);

__device__ void d_set_matrix_position(Matrix D, int i, int j, float value);
void h_set_matrix_position(Matrix D, int i, int j, float value);

__global__ void min_Q(Matrix D, Vector S, Matrix Q);

int main(int argc, char const *argv[])
{
    int N, iter ;
    float value, sum_i, sum_j;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    Matrix h_D, d_D, d_Q;
    Vector h_S, d_S;
    size_t size_D, size_S;

    if(argc != 2){
      printf("missing filename\n");
      return 0;
    }

    FILE* f = fopen(argv[1], "r");
    if(!f) return 0;

    if (fscanf(f,"%d ",&N) != 1) goto READERRO;

    size_D = (N*(N-1)/2)*sizeof(float);
    h_D.elements = (float*) malloc(size_D);
    if(h_D.elements == NULL) goto EXIT;

    size_S = N*sizeof(float);
    h_S.elements = (float*) calloc(N, sizeof(float));
    if(h_S.elements == NULL) goto EXIT;

    // Distancias
    for(int i = 0; i < N; ++i)
    {
        for (int j = 0; j < i; ++j)
        {   
            if(fscanf(f, "%f;", &value) != 1) goto READERRO;
            h_set_matrix_position(h_D, i, j, value);   
            
            sum_i = h_S.elements[i];
            sum_j = h_S.elements[j];

            h_S.elements[i] += sum_i + value;
            h_S.elements[j] += sum_j + value;
            // printf("%f, %d-%d\n", value, i, j);
        }
        
    }

    fclose(f);

    hipMalloc(&d_D.elements, size_D);
    hipMalloc(&d_S.elements, size_S);
    hipMemcpy(d_D.elements, h_D.elements, size_D, hipMemcpyHostToDevice);
    hipMemcpy(d_S.elements, h_S.elements, size_S, hipMemcpyHostToDevice);

    hipMalloc(&d_Q.elements, size_D);

    h_D.n = N;
    h_S.size = N;
    d_D.n = N;
    d_S.size = N;
    d_Q.n = N;

    // A EXECUCAO DO NJ É "LINEAR" (dependente de uma situcao anteior)
    // Somente e possivel calcular as novas distancias depois de selecionar o par
    iter = N;

    while(iter == N){

        // se iter for menor que block_size, ele nao executa nada pq vai ser zero;
        dim3 dimGrid(iter/dimBlock.x, iter/dimBlock.y);
        min_Q<<<dimGrid, dimBlock>>>(d_D, d_S, d_Q);
        hipDeviceSynchronize();
        // q_heap<<<aaaa>>>()
        // update_D<<<aaaa>>>()

        iter--;
    }

    hipFree(d_D.elements);
    hipFree(d_S.elements);
    hipFree(d_Q.elements);

    free(h_D.elements);
    free(h_S.elements);

    return 0;

    READERRO:
    printf("Erro de leitura");
    goto EXIT;

    EXIT:
    fclose(f);
    free(h_D.elements);
    free(h_S.elements);
    hipFree(d_D.elements);
    hipFree(d_S.elements);
    hipFree(d_Q.elements);
    return 1;
    
    return 0;
}

void h_set_matrix_position(Matrix D, int i, int j, float value){
    int pos;

    if(i==j)
        return;

    pos = i*(i-1)/2 + j;
    if(i < j)
        pos = j*(j-1)/2 + i;
    D.elements[pos] = value;
}

__device__ void d_set_matrix_position(Matrix D, int i, int j, float value){
    int pos;

    if(i==j)
        return;

    pos = i*(i-1)/2 + j;
    if(i < j)
        pos = j*(j-1)/2 + i;
    D.elements[pos] = value;
}

__device__ float d_get_matrix_position(Matrix D, int i, int j){
    int pos;

    if(i==j)
        return 0;

    pos = i*(i-1)/2 + j;
    if(i < j)
        pos = j*(j-1)/2 + i;
    return D.elements[pos] ;
}

// float h_get_position(Matrix D, int i, int j){

// }

__global__ void min_Q(Matrix D, Vector S, Matrix Q){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float value, d_rc;

    if((row>=D.n) && (col>=D.n)) return;
    
    d_rc = d_get_matrix_position(D, row, col);
    value = (D.n-2)*d_rc - S.elements[row] - S.elements[col];

    printf("%d - %d\n", row, col);
    d_set_matrix_position(Q, row, col, value);
}